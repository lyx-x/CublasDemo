#include "hip/hip_runtime.h"
#include "level_1.cuh"

void vectorAbsMinMax() {
	printf("---- Demo abs min-max element in a vector ----\n");
	const int n = 6;
	hipError_t cudaStat;
	hipblasStatus_t stat;
	hipblasHandle_t handle;
	float* x = new float[n];
	generateVector(x, n, 0, 100);
	printVector(x, n, "x");
	float* d_x;
	cudaStat = hipMalloc((void**)&d_x, n * sizeof(*x));
	stat = hipblasCreate(&handle);
	// Copy vector to device
	stat = hipblasSetVector(n, sizeof(*x), x, 1, d_x, 1);
	int result;
	stat = hipblasIsamax(handle, n, d_x, 1, &result);
	printf("max |x[i]|: \t%.0f\n", fabs(x[result - 1]));
	stat = hipblasIsamin(handle, n, d_x, 1, &result);
	printf("min |x[i]|: \t%.0f\n", fabs(x[result - 1]));
	hipFree(d_x);
	hipblasDestroy(handle);
	delete[] x;
}

void vectorAbsSum() {
	printf("---- Demo abs sum of a vector ---\n");
	const int n = 6;
	hipError_t cudaStat;
	hipblasStatus_t stat;
	hipblasHandle_t handle;
	float* x = new float[n];
	generateVector(x, n, 0, 100);
	printVector(x, n, "x");
	float* d_x;
	cudaStat = hipMalloc((void**)&d_x, n * sizeof(x));
	stat = hipblasCreate(&handle);
	// Copy vector to device
	stat = hipblasSetVector(n, sizeof(*x), x, 1, d_x, 1);
	float result;
	stat = hipblasSasum(handle, n, d_x, 1, &result);
	printf("sum |x[i]|: \t%.0f\n", result);
	hipFree(d_x);
	hipblasDestroy(handle);
	free(x);
}

void vectorScalar() {
	printf("---- Demo a * x + y ---\n");
}
